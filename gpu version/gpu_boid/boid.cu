#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <vector>
#include <random>
#include <iostream>
#include <chrono>
#include <fstream>


#define BOID_NUMBER 1000
#define SAVE false
#define SIGHT_RANGE 100
#define SIGHT_RANGE_SQ SIGHT_RANGE*SIGHT_RANGE
#define LENGTH 1000
#define MAX_SPEED 3.0
#define MAX_FORCE 0.6
#define STEPS 1000
#define PathIndice(boid,step) step*BOID_NUMBER+boid

using namespace std;

/**
 * \brief  Returns the square of the Euclidean distance between two vectors
 * \param  x1 | vector 1 x component
 * \param  x2 | vector 2 x component
 * \param  y1 | vector 1 y component
 * \param  y2 | vector 2 y component
 * \param  z1 | vector 1 z component
 * \param  z2 | vector 2 z component
 * \return  | euclidean distance squared between the two vectors
 */
__device__
float SquareDistance(float &x1, float &x2, float &y1, float &y2, float &z1, float &z2)
{

	return pow(x1 - x2, 2) + pow(y1 - y2, 2) + pow(z1 - z2, 2);
}

/**
 * \brief Returns the magnitude of a vector
 * \param  x | vector x component
 * \param  y | vector y component
 * \param  z | vector z component
 * \return  | magnitude of the vector
 */
__device__
float Magnitude(float &x, float &y, float &z)
{
	return sqrt(pow(x, 2) + pow(y, 2) + pow(z, 2));
}

/**
 * \brief  Returns the magnitude squared of a vector
 * \param  x | vector x component
 * \param  y | vector y component
 * \param  z | vector z component
 * \return  | magnitude squared
 */
__device__
float MagnitudeSq(float &x, float &y, float &z)
{
	return pow(x, 2) + pow(y, 2) + pow(z, 2);
}

/**
 * \brief  Updates the boids position according to the boid model.
 *		   One function which integrates cohesion,seperation and alignment
 * \param  vel_array | The array of floats that store the boids velocities
 * \param  pos_array | The array of floats that store the boids positions
 * \param  i | The index of the array where the given boids velocity/position can be found
 */
__device__
void UpdateBoid(float *vel_array, float *pos_array, int i)
{
	float pos_x = pos_array[i];
	float pos_y = pos_array[i + 1];
	float pos_z = pos_array[i + 2];

	float vel_x = vel_array[i];
	float vel_y = vel_array[i + 1];
	float vel_z = vel_array[i + 2];

	//Cohesion average velocity vector
	float cohesion_ave_vel_x = 0;
	float cohesion_ave_vel_y = 0;
	float cohesion_ave_vel_z = 0;

	//Seperation average position vector
	float seperation_ave_pos_x = 0;
	float seperation_ave_pos_y = 0;
	float seperation_ave_pos_z = 0;

	//Alignment average centre of mass vector
	float align_ave_com_x = 0;
	float align_ave_com_y = 0;
	float align_ave_com_z = 0;

	int num_boids = 0;

	for (int j = 0; j < BOID_NUMBER; j++)
	{
		float boid_pos_x = pos_array[3 * j];
		float boid_pos_y = pos_array[3 * j + 1];
		float boid_pos_z = pos_array[3 * j + 2];


		float sq_dist = SquareDistance(pos_x, boid_pos_x, pos_y, boid_pos_y, pos_z, boid_pos_z);

		if (sq_dist < SIGHT_RANGE_SQ && sq_dist != 0)
		{
			float boid_vel_x = vel_array[3 * j];
			float boid_vel_y = vel_array[3 * j + 1];
			float boid_vel_z = vel_array[3 * j + 2];

			num_boids++;

			cohesion_ave_vel_x += boid_vel_x;
			cohesion_ave_vel_y += boid_vel_y;
			cohesion_ave_vel_z += boid_vel_z;
			
			float dist = sqrt(sq_dist);
			seperation_ave_pos_x += (pos_x - boid_pos_x) / dist;
			seperation_ave_pos_y += (pos_y - boid_pos_y) / dist;
			seperation_ave_pos_z += (pos_z - boid_pos_z) / dist;
			
			align_ave_com_x += boid_pos_x;
			align_ave_com_y += boid_pos_y;
			align_ave_com_z += boid_pos_z;
		}
	}

	if (num_boids > 0)
	{
		align_ave_com_x /= num_boids;
		align_ave_com_y /= num_boids;
		align_ave_com_z /= num_boids;

		seperation_ave_pos_x /= num_boids;
		seperation_ave_pos_y /= num_boids;
		seperation_ave_pos_z /= num_boids;

		cohesion_ave_vel_x /= num_boids;
		cohesion_ave_vel_y /= num_boids;
		cohesion_ave_vel_z /= num_boids;

		//COHESION
		float cohesion_vec_magnitude = Magnitude(cohesion_ave_vel_x, cohesion_ave_vel_y, cohesion_ave_vel_z);
		cohesion_ave_vel_x = cohesion_ave_vel_x * MAX_SPEED / cohesion_vec_magnitude;
		cohesion_ave_vel_y = cohesion_ave_vel_y * MAX_SPEED / cohesion_vec_magnitude;
		cohesion_ave_vel_z = cohesion_ave_vel_z * MAX_SPEED / cohesion_vec_magnitude;

		float cohesion_correction_x = cohesion_ave_vel_x - vel_x;
		float cohesion_correction_y = cohesion_ave_vel_y - vel_y;
		float cohesion_correction_z = cohesion_ave_vel_z - vel_z;

		float cohesion_correction_magnitude = Magnitude(cohesion_correction_x, cohesion_correction_y, cohesion_correction_z);
		cohesion_correction_x = cohesion_correction_x * MAX_FORCE / cohesion_correction_magnitude;
		cohesion_correction_y = cohesion_correction_y * MAX_FORCE / cohesion_correction_magnitude;
		cohesion_correction_z = cohesion_correction_z * MAX_FORCE / cohesion_correction_magnitude;

		//SEPERATION
		float seperation_vec_magnitude_sq = MagnitudeSq(seperation_ave_pos_x, seperation_ave_pos_y, seperation_ave_pos_z);

		if (seperation_vec_magnitude_sq > 0) 
		{
			float seperation_vec_magnitude = sqrt(seperation_vec_magnitude_sq);
			seperation_ave_pos_x = seperation_ave_pos_x * MAX_SPEED / seperation_vec_magnitude;
			seperation_ave_pos_y = seperation_ave_pos_y * MAX_SPEED / seperation_vec_magnitude;
			seperation_ave_pos_z = seperation_ave_pos_z * MAX_SPEED / seperation_vec_magnitude;
		}

		float seperation_correction_x = seperation_ave_pos_x - vel_x;
		float seperation_correction_y = seperation_ave_pos_y - vel_y;
		float seperation_correction_z = seperation_ave_pos_z - vel_z;

		float seperation_correc_mag_sq = MagnitudeSq(seperation_correction_x, seperation_correction_y, seperation_correction_z);

		if (seperation_correc_mag_sq > MAX_FORCE*MAX_FORCE)
		{
			float seperation_correc_mag = sqrt(seperation_correc_mag_sq);
			seperation_correction_x = seperation_correction_x * MAX_FORCE / seperation_correc_mag;
			seperation_correction_y = seperation_correction_y * MAX_FORCE / seperation_correc_mag;
			seperation_correction_z = seperation_correction_z * MAX_FORCE / seperation_correc_mag;
		}

		//Alignment
		float vec_to_com_x = align_ave_com_x - pos_x;
		float vec_to_com_y = align_ave_com_y - pos_y;
		float vec_to_com_z = align_ave_com_z - pos_z;

		float com_vec_mag_sq = MagnitudeSq(vec_to_com_x, vec_to_com_y, vec_to_com_z);

		if (com_vec_mag_sq > 0)
		{
			float com_vec_mag = sqrt(com_vec_mag_sq);
			vec_to_com_x = vec_to_com_x * MAX_SPEED / com_vec_mag;
			vec_to_com_y = vec_to_com_y * MAX_SPEED / com_vec_mag;
			vec_to_com_z = vec_to_com_z * MAX_SPEED / com_vec_mag;
		}

		float align_correction_x = vec_to_com_x - vel_x;
		float align_correction_y = vec_to_com_y - vel_y;
		float align_correction_z = vec_to_com_z - vel_z;

		float align_correction_mag_sq = MagnitudeSq(align_correction_x, align_correction_y, align_correction_z);

		if (align_correction_mag_sq > MAX_FORCE*MAX_FORCE)
		{
			float align_correction_mag = sqrt(align_correction_mag_sq);
			align_correction_x = align_correction_x * MAX_FORCE / align_correction_mag;
			align_correction_y = align_correction_y * MAX_FORCE / align_correction_mag;
			align_correction_z = align_correction_z * MAX_FORCE / align_correction_mag;
		}

		vel_array[i] += align_correction_x + 1.05*seperation_correction_x + cohesion_correction_x;
		vel_array[i + 1] += align_correction_y + 1.05*seperation_correction_y + cohesion_correction_y;
		vel_array[i + 2] += align_correction_z + 1.05*seperation_correction_z + cohesion_correction_z;
	}

	//Update boids kinematics and impose boundary conditions
	pos_array[i] += vel_array[i];
	pos_array[i + 1] += vel_array[i + 1];
	pos_array[i + 2] += vel_array[i + 2];

	pos_array[i] = pos_array[i] > LENGTH ? 0 : pos_array[i];
	pos_array[i + 1] = pos_array[i + 1] > LENGTH ? 0 : pos_array[i + 1];
	pos_array[i + 2] = pos_array[i + 2] > LENGTH ? 0 : pos_array[i + 2];

	pos_array[i] = pos_array[i] < 0 ? LENGTH : pos_array[i];
	pos_array[i + 1] = pos_array[i + 1] < 0 ? LENGTH : pos_array[i + 1];
	pos_array[i + 2] = pos_array[i + 2] < 0 ? LENGTH : pos_array[i + 2];
}

/**
 * \brief  Main cuda kernel thar runs on the GPU updating all the boids,
 *		   distributing them to many threads.
 * \param  vel_array | Array of floats storing velocities of boids
 * \param  pos_array | Array of floats storing positions of boids
 */
__global__
void UpdateBoids(float *vel_array, float *pos_array)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < BOID_NUMBER; i += stride)
	{
		UpdateBoid(vel_array, pos_array, 3 * i);
	}
}

//Convenience structure for handling position vectors
struct vec3
{
	float x;
	float y;
	float z;
};

/**
 * \brief  Writes simulation data to a file
 * \param  name | Name of the file
 * \param  paths | Data of the simulation (boid trajectories)
 */
void WriteToFile(std::string name, std::vector<vec3> &paths)
{
	ofstream file;
	file.open(name + ".txt");

	file << "Boid Simulation Output Results:" << endl;
	file << "Time of Simulation: " << "None" << endl << endl;
	file << "Number of Boids: " << BOID_NUMBER << endl;
	file << "Size of Simulation Area: " << LENGTH << endl;
	file << "Number of Simulation Steps: " << STEPS << endl;
	file << endl;

	for (int step = 0; step < STEPS; step++)
	{
		for (int boid = 0; boid < BOID_NUMBER; boid++)
		{
			vec3 pos = paths[PathIndice(boid, step)];

			file << pos.x << ':' << pos.y << ":" << pos.z << "$";
		}
		file << "\n";
	}

	file.close();
}

int main()
{
	float* pos_array;
	float* vel_array;
	std::vector<vec3> paths(BOID_NUMBER*STEPS);

	//Allocate unified memory acessible by both host (PC) and device (GPU)
	hipMallocManaged(&pos_array, 3 * BOID_NUMBER * sizeof(float));
	hipMallocManaged(&vel_array, 3 * BOID_NUMBER * sizeof(float));

	random_device rand_dev;
	default_random_engine ran_num_gen(rand_dev());
	uniform_real_distribution<float> position_distribution(LENGTH / 4, 3 * LENGTH / 4);
	uniform_real_distribution<float> velocity_distribution(-MAX_SPEED, MAX_SPEED);

	//Assigns more threads as the boid number grows
	int blockSize = 256;
	int numBlocks = (BOID_NUMBER + blockSize - 1) / blockSize;

	for (int i = 0; i < 3 * BOID_NUMBER; i++)
	{
		pos_array[i] = position_distribution(ran_num_gen);
		vel_array[i] = velocity_distribution(ran_num_gen);
	}

	auto start = std::chrono::high_resolution_clock::now();
	for (int step = 0; step < STEPS; step++)
	{
		UpdateBoids << <numBlocks, blockSize >> > (vel_array, pos_array);
		hipDeviceSynchronize();

		for (int boid = 0; boid < BOID_NUMBER; boid++)
		{
			paths[PathIndice(boid, step)].x = pos_array[boid * 3];
			paths[PathIndice(boid, step)].y = pos_array[boid * 3 + 1];
			paths[PathIndice(boid, step)].z = pos_array[boid * 3 + 2];
		}
	}
	
	auto stop = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);

	std::cout << "Time Taken " << ": " << duration.count() / 1000.0 << "s" << endl;
	if (SAVE)
	{
		WriteToFile("gpu_test", paths);
	}
	hipFree(vel_array);
	hipFree(pos_array);

	return 0;
}




